#include "hip/hip_runtime.h"
/*****************************************************************************/
/*IMPORTANT:  READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.         */
/*By downloading, copying, installing or using the software you agree        */
/*to this license.  If you do not agree to this license, do not download,    */
/*install, copy or use the software.                                         */
/*                                                                           */
/*                                                                           */
/*Copyright (c) 2005 Northwestern University                                 */
/*All rights reserved.                                                       */

/*Redistribution of the software in source and binary forms,                 */
/*with or without modification, is permitted provided that the               */
/*following conditions are met:                                              */
/*                                                                           */
/*1       Redistributions of source code must retain the above copyright     */
/*        notice, this list of conditions and the following disclaimer.      */
/*                                                                           */
/*2       Redistributions in binary form must reproduce the above copyright   */
/*        notice, this list of conditions and the following disclaimer in the */
/*        documentation and/or other materials provided with the distribution.*/
/*                                                                            */
/*3       Neither the name of Northwestern University nor the names of its    */
/*        contributors may be used to endorse or promote products derived     */
/*        from this software without specific prior written permission.       */
/*                                                                            */
/*THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS ``AS    */
/*IS'' AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED      */
/*TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY, NON-INFRINGEMENT AND         */
/*FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL          */
/*NORTHWESTERN UNIVERSITY OR ITS CONTRIBUTORS BE LIABLE FOR ANY DIRECT,       */
/*INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES          */
/*(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR          */
/*SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)          */
/*HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,         */
/*STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN    */
/*ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE             */
/*POSSIBILITY OF SUCH DAMAGE.                                                 */
/******************************************************************************/
/*************************************************************************/
/**   File:         kmeans_clustering.c                                 **/
/**   Description:  Implementation of regular k-means clustering        **/
/**                 algorithm                                           **/
/**   Author:  Wei-keng Liao                                            **/
/**            ECE Department, Northwestern University                  **/
/**            email: wkliao@ece.northwestern.edu                       **/
/**                                                                     **/
/**   Edited by: Jay Pisharath                                          **/
/**              Northwestern University.                               **/
/**                                                                     **/
/**   ================================================================  **/
/**																		**/
/**   Edited by: Sang-Ha  Lee											**/
/**				 University of Virginia									**/
/**																		**/
/**   Description:	No longer supports fuzzy c-means clustering;	 	**/
/**					only regular k-means clustering.					**/
/**					Simplified for main functionality: regular k-means	**/
/**					clustering.											**/
/**                                                                     **/
/*************************************************************************/
// To solve linking problems
extern "C"
{
#include "kmeans.h"
}

#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include "kmeans.h"
#include <omp.h>

#define RANDOM_MAX 2147483647

#ifndef FLT_MAX
#define FLT_MAX 3.40282347e+38
#endif

// CUDA Config
#define THREADS_PER_BLOCK 32
#define WARP_SIZE 32
#define TOTAL_THREAD_LIMIT 1024

#define THREAD_LIMIT_X 1024
#define THREAD_LIMIT_Y 1024
#define THREAD_LIMIT_Z 64

#define BLOCK_LIMIT_X 2147483647
#define BLOCK_LIMIT_Y 65535
#define BLOCK_LIMIT_Z 65535

#define checkError(ans)                       \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "Device error: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

extern double wtime(void);

__constant__ int d_nfeatures;
__constant__ int d_npoints;
__constant__ int d_nclusters;
__constant__ float d_threshold;

/* ==================== Host util functions ==================== */

int updiv(int threads_per_block, int N)
{
    return (N + threads_per_block - 1) / threads_per_block;
}

int config_check(int npoints, int nfeatures, int nclusters)
{
    if (nfeatures > THREAD_LIMIT_X)
        return 1;

    if (nclusters > THREAD_LIMIT_Y)
        return 1;

    if (nfeatures * nclusters > TOTAL_THREAD_LIMIT)
        return 1;

    int threads_per_block_x = TOTAL_THREAD_LIMIT / nfeatures;

    if (nfeatures > THREAD_LIMIT_Y)
        return 1;

    if (threads_per_block_x > THREAD_LIMIT_X)
        return 1;

    if (nfeatures * threads_per_block_x > TOTAL_THREAD_LIMIT)
        return 1;

    if (updiv(threads_per_block_x, npoints) > BLOCK_LIMIT_X)
        return 1;

    if (updiv(THREADS_PER_BLOCK, npoints) > BLOCK_LIMIT_X)
        return 1;

    if (THREADS_PER_BLOCK > TOTAL_THREAD_LIMIT)
        return 1;

    return 0;
}

/* ==================== Init functions ==================== */

__global__ void init_cluster_centers(float *d_clusters, float *d_feature)
{
    int cluster = blockIdx.y * blockDim.y + threadIdx.y;
    int feature = blockIdx.x * blockDim.x + threadIdx.x;

    if (cluster < d_nclusters && feature < d_nfeatures)
    {
        d_clusters[d_nfeatures * cluster + feature] = d_feature[d_nfeatures * cluster + feature];
    }
}

__global__ void init_membership(int *d_membership)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < d_npoints)
    {
        d_membership[tid] = -1;
    }
}

__global__ void reset_aux_data(int *d_new_centers_len, float *d_new_centers, float *d_delta)
{
    int cluster = blockIdx.y * blockDim.y + threadIdx.y;
    int feature = blockIdx.x * blockDim.x + threadIdx.x;

    if (cluster < d_nclusters && feature < d_nfeatures)
    {
        // Only 1 thread will reset delta
        if (cluster == 0 && feature == 0)
            *d_delta = 0;

        // Only 1 thread per cluster will reset len
        if (feature == 0)
            d_new_centers_len[cluster] = 0;

        d_new_centers[cluster * d_nfeatures + feature] = 0.0;
    }
}

/* ==================== Main computational functions ==================== */

__global__ void assign_membership(float *d_feature, float *d_clusters, int *d_membership, float *d_new_centers, int *d_new_centers_len, float *d_delta, int optimized)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;
    int index, j, i;
    float dist, min_dist;
    float aux1, aux2;

    extern __shared__ float s_clusters[];
    __shared__ float s_delta;

    // Load to share memory
    if (optimized) // Number of threads is enough to load all data (bigger than d_nclusters * d_nfeatures)
    {
        if (local_tid < d_nclusters * d_nfeatures)
            s_clusters[local_tid] = d_clusters[local_tid];
    }
    else
    {
        if (local_tid < d_nclusters)
        {
            for (j = 0; j < d_nfeatures; j++)
            {
                s_clusters[threadIdx.x * d_nfeatures + j] = d_clusters[threadIdx.x * d_nfeatures + j];
            }
        }
    }

    if (local_tid == 0)
    {
        s_delta = 0;
    }

    __syncthreads();

    if (tid < d_npoints)
    {

        /* ========== find_nearest_point function start ========== */
        min_dist = FLT_MAX;
        for (i = 0; i < d_nclusters; i++)
        {

            /* ========== euclid_dist_2 function start ========== */
            dist = 0;

            for (j = 0; j < d_nfeatures; j++)
            {
                aux1 = d_feature[tid * d_nfeatures + j];
                aux2 = s_clusters[i * d_nfeatures + j];

                dist += (aux1 - aux2) * (aux1 - aux2);
            }

            /* ========== euclid_dist_2 function end ========== */

            if (dist < min_dist)
            {
                min_dist = dist;
                index = i;
            }
        }
        /* ========== find_nearest_point function end ========== */

        /* if membership changes, increase delta by 1 */
        if (s_delta < d_threshold && d_membership[tid] != index)
            atomicAdd(&s_delta, 1.0f);

        d_membership[tid] = index;
    }

    __syncthreads();

    // Store in main memory
    if (local_tid == 0)
    {
        atomicAdd(d_delta, s_delta);
    }
}

// Kernel to sum cluster centers
__global__ void sum_clusters(float *d_feature, int *d_membership, float *d_new_centers, int *d_new_centers_len)
{
    int point = blockIdx.x * blockDim.x + threadIdx.x;
    int local_point = threadIdx.x;
    int feature = blockIdx.y * blockDim.y + threadIdx.y;
    int local_feature = threadIdx.y;
    int index;
    float *s_new_centers;
    int *s_new_centers_len;

    extern __shared__ char shared[];

    s_new_centers = (float *)shared;
    s_new_centers_len = (int *)(shared + d_nclusters * d_nfeatures * sizeof(float));

    // Init shared memory

    // Only nclusters
    if (local_feature == 0 && local_point < d_nclusters)
        s_new_centers_len[local_point] = 0;

    if (local_point < d_nclusters && local_feature < d_nfeatures)
        s_new_centers[local_point * d_nfeatures + local_feature] = 0.0f;

    __syncthreads();

    if (point < d_npoints && feature < d_nfeatures)
    {
        index = d_membership[point];

        // Only 1 thread per point
        if (feature == 0)
            atomicAdd(s_new_centers_len + index, 1);

        atomicAdd(s_new_centers + index * d_nfeatures + feature, d_feature[point * d_nfeatures + feature]);
    }

    __syncthreads();

    // Store in main memory
    if (local_feature == 0 && local_point < d_nclusters)
        atomicAdd(&d_new_centers_len[local_point], s_new_centers_len[local_point]);

    if (local_point < d_nclusters && local_feature < d_nfeatures)
        atomicAdd(&d_new_centers[local_point * d_nfeatures + local_feature], s_new_centers[local_point * d_nfeatures + local_feature]);
}

// Kernel to divide each new cluster center
__global__ void divide_clusters(float *d_clusters, float *d_new_centers, int *d_new_centers_len)
{
    int cluster = blockIdx.y * blockDim.y + threadIdx.y;
    int feature = blockIdx.x * blockDim.x + threadIdx.x;
    int len;

    extern __shared__ int s_new_centers_len[];

    if (feature == 0) {
        s_new_centers_len[cluster] = d_new_centers_len[cluster];
    }

    __syncthreads();

    if (cluster < d_nclusters && feature < d_nfeatures)
    {
        len = s_new_centers_len[cluster];
        if (len > 0)
        {
            d_clusters[cluster * d_nfeatures + feature] = d_new_centers[cluster * d_nfeatures + feature] / len;
        }
    }
}

/*----< kmeans_clustering() >---------------------------------------------*/
// To solve linking problems
extern "C" float **kmeans_clustering(float **feature, /* in: [npoints][nfeatures] */
                                     int nfeatures,
                                     int npoints,
                                     int nclusters,
                                     float threshold,
                                     int *membership) /* out: [npoints] */
{

    int i;
    float delta;
    float **clusters; /* out: [nclusters][nfeatures] */
    int optimized = 0;

    /* =============== Device vars =============== */
    int *d_membership;
    int *d_new_centers_len; /* [nclusters]: no. of points in each cluster */
    float *d_feature;
    float *d_delta;
    float *d_clusters;    /* out: [nclusters][nfeatures] */
    float *d_new_centers; /* [nclusters][nfeatures] */

    if (config_check(npoints, nfeatures, nclusters))
    {
        printf("Configuration of (npoints, nfeatures, nclusters) not currently supported (exceeds block and thread limits)\n");
        exit(1);
    }
    dim3 clusters_gridDist(1, 1, 1);
    dim3 clusters_blockDist(nfeatures, nclusters, 1);

    dim3 points_gridDist(updiv(TOTAL_THREAD_LIMIT / nfeatures, npoints), 1, 1);
    dim3 points_blockDist(TOTAL_THREAD_LIMIT / nfeatures, nfeatures, 1);

    checkError(hipMalloc((void **)&d_membership, npoints * sizeof(int)));
    checkError(hipMalloc((void **)&d_new_centers_len, nclusters * sizeof(int)));
    checkError(hipMalloc((void **)&d_feature, npoints * nfeatures * sizeof(float)));
    checkError(hipMalloc((void **)&d_delta, sizeof(float)));
    checkError(hipMalloc((void **)&d_clusters, nclusters * nfeatures * sizeof(float)));
    checkError(hipMalloc((void **)&d_new_centers, nclusters * nfeatures * sizeof(float)));

    hipMemcpy2DAsync(d_feature, nfeatures * sizeof(float), feature[0], nfeatures * sizeof(float), nfeatures * sizeof(float), npoints, hipMemcpyHostToDevice);

    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_nfeatures), &nfeatures, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_npoints), &npoints, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_nclusters), &nclusters, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_threshold), &threshold, sizeof(float));

    /* =============== allocate space for returning variable clusters[] =============== */
    clusters = (float **)malloc(nclusters * sizeof(float *));
    clusters[0] = (float *)malloc(nclusters * nfeatures * sizeof(float));

    if (clusters == NULL || clusters[0] == NULL)
    {
        printf("Couldn't allocate clusters, exiting.");
        exit(1);
    }

    for (i = 1; i < nclusters; i++)
        clusters[i] = clusters[i - 1] + nfeatures;

    checkError(hipDeviceSynchronize()); // Check for errors in the Async calls

    /* =============== initialization  =============== */

    init_cluster_centers<<<clusters_gridDist, clusters_blockDist>>>(d_clusters, d_feature);
    checkError(hipGetLastError());

    init_membership<<<updiv(THREADS_PER_BLOCK, npoints), THREADS_PER_BLOCK>>>(d_membership);
    checkError(hipGetLastError());

    if (THREADS_PER_BLOCK >= nfeatures * nclusters)
        optimized = 1;
    /* =============== Main computation part  =============== */
    do
    {
        /* =============== Set auxiliary data structures to 0  =============== */
        reset_aux_data<<<clusters_gridDist, clusters_blockDist>>>(d_new_centers_len, d_new_centers, d_delta);
        checkError(hipGetLastError());

        /* =============== assign membership to each point  =============== */
        assign_membership<<<updiv(THREADS_PER_BLOCK, npoints), THREADS_PER_BLOCK, nclusters * nfeatures * sizeof(float)>>>(d_feature, d_clusters, d_membership, d_new_centers, d_new_centers_len, d_delta, optimized);
        checkError(hipGetLastError());

        /* =============== calculate the centers (average)  =============== */
        sum_clusters<<<points_gridDist, points_blockDist, nclusters * nfeatures * sizeof(float) + nclusters * sizeof(int)>>>(d_feature, d_membership, d_new_centers, d_new_centers_len);
        checkError(hipGetLastError());
        divide_clusters<<<clusters_gridDist, clusters_blockDist, nclusters * sizeof(int)>>>(d_clusters, d_new_centers, d_new_centers_len);
        checkError(hipGetLastError());

        checkError(hipMemcpy(&delta, d_delta, sizeof(float), hipMemcpyDeviceToHost)); // Also returns errors in the kernels

    } while (delta > threshold);

    /* =============== copy final results to host =============== */
    for (i = 0; i < nclusters; i++)
    {
        hipMemcpyAsync(clusters[i], d_clusters + i * nfeatures, nfeatures * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipMemcpyAsync(membership, d_membership, npoints * sizeof(int), hipMemcpyDeviceToHost);

    checkError(hipDeviceSynchronize()); // Check for errors in the Async calls

    /* =============== free memory =============== */
    checkError(hipFree(d_membership));
    checkError(hipFree(d_new_centers_len));
    checkError(hipFree(d_feature));
    checkError(hipFree(d_delta));
    checkError(hipFree(d_clusters));
    checkError(hipFree(d_new_centers));

    return clusters;
}
